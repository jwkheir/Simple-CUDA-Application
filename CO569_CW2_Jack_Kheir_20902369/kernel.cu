#include "hip/hip_runtime.h"

#include <stdio.h>
#include <fstream>
#include <iostream>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""

using namespace std;

const int DATA_SET_SIZE = 18;
const int MOVES = 3;
const int PERMUTATIONS = MOVES * MOVES;
int NAB[PERMUTATIONS];
int data[DATA_SET_SIZE];

const int N = DATA_SET_SIZE;

// Generate a random set of integers each being in the range 0 to MOVES - 1
// Save numbers to file to ensure tests can be repeated.
void GenerateData(){
	ofstream out("data.dat", ios::out | ios::binary);
	for (int n = 0; n < DATA_SET_SIZE; n++){
		int i = rand() % MOVES;
		out.write((char *)&i, sizeof(i));
	}
	out.close();
}

// Populate data array with contents from file
void GetData(){
	ifstream in("data.dat", ios::in | ios::binary);
	for (int n = 0; n < DATA_SET_SIZE; n++){
		in.read((char *)&data[n], sizeof(int));
	}
	in.close();
}

/*
Intialise NAB pointer allocated on GPU device
*/
__global__ void InitialiseNAB(int *NAB){
	int i = blockIdx.x;
	NAB[i] = 0;
}

/*
This device function is called from the global PopulateNABblocks/PopulateNABthreads kernel
*/
__device__ int GetIndex(int firstMove, int secondMove){
	if (firstMove == 0 && secondMove == 0) return 0;
	if (firstMove == 0 && secondMove == 1) return 1;
	if (firstMove == 0 && secondMove == 2) return 2;
	if (firstMove == 1 && secondMove == 0) return 3;
	if (firstMove == 1 && secondMove == 1) return 4;
	if (firstMove == 1 && secondMove == 2) return 5;
	if (firstMove == 2 && secondMove == 0) return 6;
	if (firstMove == 2 && secondMove == 1) return 7;
	if (firstMove == 2 && secondMove == 2) return 8;
}

__global__ void PopulateNABblocks(int *NAB, int *data){
	int index = blockIdx.x;
	int i;
	int previous = data[index];
	int next = data[index + 1];

	if (index < DATA_SET_SIZE)
	{
		i = GetIndex(previous, next);
	}
	NAB[i]++;
}

__global__ void PopulateNABThreads(int *NAB, int *data){
	int index = threadIdx.x;
	int i;
	int previous = data[index];
	if (index < DATA_SET_SIZE)
	{
		i = GetIndex(previous, data[index + 1]);
	}
	NAB[i]++;
}

void DisplayNAB(){
	cout << endl;
	for (int n = 0; n < PERMUTATIONS; n++){
		cout << "Index " << n << " : " << NAB[n] << endl;
	}
}

int main(){
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	int maxThreads = prop.maxThreadsPerBlock;

	srand(time(NULL));
	GenerateData();
	GetData();
	
	int *dev_NAB;
	int *dev_data;

	hipMalloc((void**)&dev_NAB, PERMUTATIONS * sizeof(int));
	hipMalloc((void**)&dev_data, DATA_SET_SIZE * sizeof(int));

	hipMemcpy(dev_data, &data, DATA_SET_SIZE * sizeof(int), hipMemcpyHostToDevice);

	InitialiseNAB << <PERMUTATIONS, 1 >> >(dev_NAB);
	PopulateNABblocks << <DATA_SET_SIZE, 1 >> >(dev_NAB, dev_data);
	//PopulateNABThreads << <1, DATA_SET_SIZE>> >(dev_NAB, dev_data);
	hipMemcpy(NAB, dev_NAB, PERMUTATIONS * sizeof(int), hipMemcpyDeviceToHost);
	DisplayNAB();

	hipFree(dev_NAB);
	hipFree(dev_data);
	getchar();
	return 0;
}